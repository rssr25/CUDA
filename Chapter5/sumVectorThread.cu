#include "hip/hip_runtime.h"
#include<hip/hip_runtime_api.h>
#include<stdio.h>

#define N 10

__global__ void add(int *a, int *b, int *c){

    int tid = threadIdx.x;
    if(tid < N) c[tid] = a[tid] + b[tid];
}
int main(void){

    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;


    //allocate memory
    checkCudaErrors(hipMalloc((void**)&dev_a, N*sizeof(int)));
    checkCudaErrors(hipMalloc((void**)&dev_b, N*sizeof(int)));
    checkCudaErrors(hipMalloc((void**)&dev_c, N*sizeof(int)));

    //initialize data
    for(int i = 0; i < N; i++){

        a[i] = i;
        b[i] = i*10;
    }

    //get these arrays to the device
    checkCudaErrors(hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice));
    
    //run the kernel
    add<<<1, N>>>(dev_a, dev_b, dev_c);

    //copy result from device to host
    checkCudaErrors(hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost));

    //show results
    for(int i=0; i < N; i++){

        printf("%d, ", c[i]);
    }

    printf("\n");
    
    //free the memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    
    return 0;
}