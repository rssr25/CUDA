#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime_api.h>

__global__ void add(int a, int b, int *c){

    *c = a+b;
}

int main(void){

    int c;
    int *dev_c;

    checkCudaErrors(hipMalloc((void**)&dev_c, sizeof(int)));
    add<<<1, 1>>>(2, 7, dev_c);
    checkCudaErrors(hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost));

    printf("2+7 = %d\n", c);
    hipFree(dev_c);
    return 0;
}