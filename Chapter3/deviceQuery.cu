#include<hip/hip_runtime_api.h>
#include<stdio.h>
int main(void){

    hipDeviceProp_t prop;

    int count;
    checkCudaErrors(hipGetDeviceCount(&count));
    for(int i = 0; i < count ; i++){
        checkCudaErrors(hipGetDeviceProperties(&prop, i));
        printf(prop.name);
    }
    printf("%d", count);
    return 0;

}