#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime_api.h>

#define N 10

__global__ void add(int *a, int *b, int *c){

    int tid = blockIdx.x;
    if(tid < N){
        c[tid] = a[tid] + b[tid];
    }
    
}

int main(void){

//define variables
int a[N], b[N], c[N];
int *dev_a, *dev_b, *dev_c;

//allocate memory to the arrays that go to the device
checkCudaErrors(hipMalloc( (void**)&dev_a, N* sizeof(int) ));
checkCudaErrors(hipMalloc( (void**)&dev_b, N* sizeof(int) ));
checkCudaErrors(hipMalloc( (void**)&dev_c, N* sizeof(int) ));

//filling the arrays a and b with data on the host
for(int i = 0; i < N; i++){
    a[i] = i;
    b[i] = i * 10;
}

//copy this data to the device
checkCudaErrors(hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice));
checkCudaErrors(hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice));

//run the kernel
add<<<N, 1>>>(dev_a, dev_b, dev_c);

//get the data from the device
checkCudaErrors(hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost));

//display the data
for(int j = 0; j < N; j++){
    printf("%d\t", c[j]);
}
printf("\n");

//free the memory
hipFree(dev_c);
hipFree(dev_b);
hipFree(dev_a);

return 0;
}